#include "hip/hip_runtime.h"
#include <stdio.h> //NULL, printf
#include <stdlib.h> //srand, rand
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include ""
#include <sstream>
#include <iostream>
#include <string.h>
#include <time.h>
#include "matVec_gpu.cuh"
#include "../cpu/matVec_cpu.h"
//nvcc -arch=sm_37
//CUDA kernels

void __global__ multiply(double * in1, unsigned int * rows1, unsigned int * cols1, double * in2, unsigned int * rows2, unsigned int * cols2, double * output){
    const unsigned int bid = blockIdx.x //1D
        + blockIdx.y * gridDim.x //2D
        + gridDim.x * gridDim.y * blockIdx.z; //3D
    const unsigned int threadsPerBlock = blockDim.x
        *blockDim.y //2D
        *blockDim.z; //3D
    const unsigned int tid = threadIdx.x //1D
        +threadIdx.y*blockDim.x //2D
        +blockDim.x*blockDim.x*threadIdx.z; //3D
    const unsigned int gid = bid * threadsPerBlock + tid;
    const unsigned int r = blockIdx.y * blockDim.y + threadIdx.y; // the row of M1
    const unsigned int c = blockIdx.x * blockDim.x + threadIdx.x; // the col of M2
    //printf("thread(%d,%d,%d), block(%d,%d,%d), bid=%d, gid=%d, %f * %f\n",threadIdx.x,threadIdx.y,threadIdx.z,
    //    blockIdx.x,blockIdx.y,blockIdx.z,bid,gid,in1[gid],in2[gid]);
    double sum = 0;
    //printf("gid:%i, %i %i %i %i\n",gid,*rows1, *cols1, *rows2, *cols2);
    if (*cols1 == *rows2){
        printf("row: %i \n",r);
        for (int i = 0; i < *rows1; i++){
            sum += in1[r * *cols1 + i] * in2[i* *cols2 + c];
        }
        output[r**cols2+c] = sum;
    }
    else{
        printf("MATRICIES CANNOT BE MULTIPLED, INVALID SIZES");
    }
}

void __global__ scale(double * input, double * scalar, double * output){
    const unsigned int bid = blockIdx.x //1D
        + blockIdx.y * gridDim.x //2D
        + gridDim.x * gridDim.y * blockIdx.z; //3D
    const unsigned int threadsPerBlock = blockDim.x
        *blockDim.y //2D
        *blockDim.z; //3D
    const unsigned int tid = threadIdx.x //1D
        +threadIdx.y*blockDim.x //2D
        +blockDim.x*blockDim.x*threadIdx.z; //3D
    const unsigned int gid = bid * threadsPerBlock + tid;
    //printf("thread(%d,%d,%d), block(%d,%d,%d), bid=%d, gid=%d, value=%f\n",threadIdx.x,threadIdx.y,threadIdx.z,
    //    blockIdx.x,blockIdx.y,blockIdx.z,bid,gid,input[gid]);
    output[gid] = input[gid] * *scalar;
    printf("%f = %f * %f\n",output[gid], input[gid], *scalar);
}

void __global__ print(double * input){
    const unsigned int bid = blockIdx.x //1D
        + blockIdx.y * gridDim.x //2D
        + gridDim.x * gridDim.y * blockIdx.z; //3D
    const unsigned int threadsPerBlock = blockDim.x
        *blockDim.y //2D
        *blockDim.z; //3D
    const unsigned int tid = threadIdx.x //1D
        +threadIdx.y*blockDim.x //2D
        +blockDim.x*blockDim.x*threadIdx.z; //3D
    const unsigned int gid = bid * threadsPerBlock + tid;
    //printf("thread(%d,%d,%d), block(%d,%d,%d), bid=%d, gid=%d, value=%f\n",threadIdx.x,threadIdx.y,threadIdx.z,
    //    blockIdx.x,blockIdx.y,blockIdx.z,bid,gid,input[gid]);
    printf("%f\n",input[gid]);
}

/*
void __global__ assignment(double * in1, double * in2){
    const unsigned int bid = blockIdx.x //1D
        + blockIdx.y * gridDim.x //2D
        + gridDim.x * gridDim.y * blockIdx.z; //3D
    const unsigned int threadsPerBlock = blockDim.x
        *blockDim.y //2D
        *blockDim.z; //3D
    const unsigned int tid = threadIdx.x //1D
        +threadIdx.y*blockDim.x //2D
        +blockDim.x*blockDim.x*threadIdx.z; //3D
    const unsigned int gid = bid * threadsPerBlock + tid;
    printf("thread(%d,%d,%d), block(%d,%d,%d), bid=%d, gid=%d, in1=%f, in2=%f\n",threadIdx.x,threadIdx.y,threadIdx.z,
        blockIdx.x,blockIdx.y,blockIdx.z,bid,gid,in1[gid],in2[gid]);
        in1[gid] = in2[gid];
    
    
}

void __global__ subtract(double * in1, double * in2, double * out){
    const unsigned int bid = blockIdx.x //1D
        + blockIdx.y * gridDim.x //2D
        + gridDim.x * gridDim.y * blockIdx.z; //3D
    const unsigned int threadsPerBlock = blockDim.x
        *blockDim.y //2D
        *blockDim.z; //3D
    const unsigned int tid = threadIdx.x //1D
        +threadIdx.y*blockDim.x //2D
        +blockDim.x*blockDim.x*threadIdx.z; //3D
    const unsigned int gid = bid * threadsPerBlock + tid;
    printf("thread(%d,%d,%d), block(%d,%d,%d), bid=%d, gid=%d, in1=%f, in2=%f\n",threadIdx.x,threadIdx.y,threadIdx.z,
        blockIdx.x,blockIdx.y,blockIdx.z,bid,gid,in1[gid],in2[gid]);
    out[gid] = in1[gid] - in2[gid];
}

void __global__ add(double * in1, double * in2, double * out){
    const unsigned int bid = blockIdx.x //1D
        + blockIdx.y * gridDim.x //2D
        + gridDim.x * gridDim.y * blockIdx.z; //3D
    const unsigned int threadsPerBlock = blockDim.x
        *blockDim.y //2D
        *blockDim.z; //3D
    const unsigned int tid = threadIdx.x //1D
        +threadIdx.y*blockDim.x //2D
        +blockDim.x*blockDim.x*threadIdx.z; //3D
    const unsigned int gid = bid * threadsPerBlock + tid;
    printf("thread(%d,%d,%d), block(%d,%d,%d), bid=%d, gid=%d, in1=%f, in2=%f\n",threadIdx.x,threadIdx.y,threadIdx.z,
        blockIdx.x,blockIdx.y,blockIdx.z,bid,gid,in1[gid],in2[gid]);
    out[gid] = in1[gid] + in2[gid];
}*/

//Operator overloads
Vector_GPU Vector_GPU::operator*(Vector_GPU &v){
    printf("MATMULT\n");
    Vector_GPU out(this->h_rows,v.h_columns);
    dim3 grid(1,1,1);
    dim3 block(out.h_rows,out.h_columns,1);
    multiply <<<grid,block>>> (this->d_mat, this->d_rows, this->d_columns, v.d_mat,  v.d_rows, v.d_columns, out.d_mat);
    return out;
}

Vector_GPU Vector_GPU::operator*(double h_i){
    printf("scale\n");
    Vector_GPU out(this->h_rows,this->h_columns);
    dim3 grid(1,1,1);
    dim3 block(this->h_rows,this->h_columns,1);
    double *d_i;
    hipMalloc((void**)&d_i,sizeof(double));
    hipMemcpy(d_i,&h_i,sizeof(double),hipMemcpyHostToDevice);
    
    scale <<<grid,block>>> (this->d_mat,d_i,out.d_mat);
    
    return out;
}

void Vector_GPU::printmat(){
    dim3 grid(1,1,1);
    printf("PRINTING\n");
    dim3 block(this->h_rows,this->h_columns,1);
    
    print <<<grid,block>>> (this->d_mat);
};

Vector_CPU Vector_GPU::matDeviceToHost(){
    double* out;
    hipMemcpy(out,this->d_mat,sizeof(double)*this->h_columns*this->h_rows,hipMemcpyDeviceToDevice);
    Vector_CPU v_cpu(this->h_rows,this->h_columns,*out);
    return v_cpu;
};

/*
Vector_GPU& Vector_GPU::operator=(const Vector_GPU &v){
    printf("Assignment operator called\n");
    this->rows = v.rows;O
    this->columns = v.columns;
    dim3 grid(1,1,1);
    dim3 block(v.rows * v.columns,1,1);
    if (columns == v.rows){
        assignment <<<grid,block>>> (this->d_mat,v.d_mat);
        
    }
    else{
        printf("ARRAYS ARE NOT THE SAME SIZE, canot perform assignment operation\n");
    }
    return *this;
}

Vector_GPU Vector_GPU::operator-(const Vector_GPU &v){
    Vector_GPU out(this->rows,this->columns);
    dim3 grid(1,1,1);
    dim3 block(v.rows * v.columns,1,1);
    if (columns == v.rows){
        subtract <<<grid,block>>> (this->d_mat,v.d_mat,out.d_mat);
        
    }
    else{
        printf("ARRAYS ARE NOT THE SAME SIZE, canot perform operation\n");
    }
    return out;   
}

Vector_GPU Vector_GPU::operator+(const Vector_GPU &v){
    Vector_GPU out(this->rows,this->columns);
    dim3 grid(1,1,1);
    dim3 block(v.rows * v.columns,1,1);
    if (columns == v.rows){
        add <<<grid,block>>> (this->d_mat,v.d_mat,out.d_mat);
        
    }
    else{
        printf("ARRAYS ARE NOT THE SAME SIZE, canot perform operation\n");
    }
    return out;   
}*/