#include "mat_cusparse.cuh"
#include <assert.h>
Vector_GPU Matrix_cuSPARSE::operator*(Vector_GPU &v) {
  Vector_GPU out(this->h_rows, 1);
  hipsparseDnVecDescr_t rhs_desc, out_desc;
  cusparseErrCheck(hipsparseCreateDnVec(&rhs_desc, out.getRows(), v.d_mat, HIP_R_64F));
  cusparseErrCheck(hipsparseCreateDnVec(&out_desc, out.getRows(), out.d_mat, HIP_R_64F));
  cusparseErrCheck(hipsparseSpMV_bufferSize(spHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &ONE, this->d_matDescr_sparse, rhs_desc, &ZERO, out_desc,
                                           HIP_R_64F, HIPSPARSE_MV_ALG_DEFAULT, &bufferSize));
  cudaErrCheck(hipMalloc(&dBuffer, this->bufferSize));
  cusparseErrCheck(hipsparseSpMV(spHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &ONE, this->d_matDescr_sparse, rhs_desc, &ZERO, out_desc, HIP_R_64F,
                                HIPSPARSE_MV_ALG_DEFAULT, dBuffer));

  return out;
};
Vector_CUBLAS Matrix_cuSPARSE::operator*(Vector_CUBLAS &v) {
  Vector_CUBLAS out(this->h_rows, 1);
  hipsparseDnVecDescr_t rhs_desc, out_desc;
  cusparseErrCheck(hipsparseCreateDnVec(&rhs_desc, out.getRows(), v.d_mat, HIP_R_64F));
  cusparseErrCheck(hipsparseCreateDnVec(&out_desc, out.getRows(), out.d_mat, HIP_R_64F));
  cusparseErrCheck(hipsparseSpMV_bufferSize(spHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &ONE, this->d_matDescr_sparse, rhs_desc, &ZERO, out_desc,
                                           HIP_R_64F, HIPSPARSE_MV_ALG_DEFAULT, &bufferSize));
  cudaErrCheck(hipMalloc(&dBuffer, this->bufferSize));
  cusparseErrCheck(hipsparseSpMV(spHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &ONE, this->d_matDescr_sparse, rhs_desc, &ZERO, out_desc, HIP_R_64F,
                                HIPSPARSE_MV_ALG_DEFAULT, dBuffer));

  return out;
};