#include "mat_cusparse.cuh"
#include <assert.h>
Vector_GPU Matrix_cuSPARSE::operator*(Vector_GPU &v) {
  Vector_GPU out(this->h_rows, 1);
  hipsparseDnVecDescr_t rhs_desc, out_desc;
  cusparseErrCheck(hipsparseCreateDnVec(&rhs_desc, out.getRows(), v.d_mat, HIP_R_64F));
  cusparseErrCheck(hipsparseCreateDnVec(&out_desc, out.getRows(), out.d_mat, HIP_R_64F));
  cusparseErrCheck(hipsparseSpMV_bufferSize(spHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &ONE, this->d_matDescr_sparse, rhs_desc, &ZERO, out_desc,
                                           HIP_R_64F, HIPSPARSE_MV_ALG_DEFAULT, &bufferSize));
  // cudaErrCheck(hipMalloc(&dBuffer, this->bufferSize));
  cusparseErrCheck(hipsparseSpMV(spHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &ONE, this->d_matDescr_sparse, rhs_desc, &ZERO, out_desc, HIP_R_64F,
                                HIPSPARSE_MV_ALG_DEFAULT, dBuffer));

  return out;
};
Vector_CUBLAS Matrix_cuSPARSE::operator*(Vector_CUBLAS &v) {
  Vector_CUBLAS out(this->h_rows, 1);
  hipsparseDnVecDescr_t rhs_desc, out_desc;
  cusparseErrCheck(hipsparseCreateDnVec(&rhs_desc, out.getRows(), v.d_mat, HIP_R_64F));
  cusparseErrCheck(hipsparseCreateDnVec(&out_desc, out.getRows(), out.d_mat, HIP_R_64F));
  cusparseErrCheck(hipsparseSpMV_bufferSize(spHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &ONE, this->d_matDescr_sparse, rhs_desc, &ZERO, out_desc,
                                           HIP_R_64F, HIPSPARSE_MV_ALG_DEFAULT, &bufferSize));
  // cudaErrCheck(hipMalloc(&dBuffer, this->bufferSize));
  cusparseErrCheck(hipsparseSpMV(spHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &ONE, this->d_matDescr_sparse, rhs_desc, &ZERO, out_desc, HIP_R_64F,
                                HIPSPARSE_MV_ALG_DEFAULT, dBuffer));

  return out;
};
Matrix_cuSPARSE Matrix_cuSPARSE::transpose() {
  Matrix_cuSPARSE out(this->h_columns, this->h_rows);
  out.h_nnz = this->h_nnz;
  printf("%d\n", out.h_nnz);
  cudaErrCheck(hipMalloc((void **)&out.d_csr_columns, out.h_nnz * sizeof(int)));
  cudaErrCheck(hipMalloc((void **)&out.d_csr_values, out.h_nnz * sizeof(double)));
  cudaErrCheck(hipMemset(out.d_csr_columns, ZERO, out.h_nnz * sizeof(int)));
  cudaErrCheck(hipMemset(out.d_csr_values, ZERO, out.h_nnz * sizeof(double)));
  cusparseErrCheck(hipsparseCsr2cscEx2_bufferSize(spHandle, this->h_rows, this->h_columns, this->h_nnz, this->d_csr_values, this->d_csr_offsets,
                                                 this->d_csr_columns, out.d_csr_values, out.d_csr_offsets, out.d_csr_columns, HIP_R_64F,
                                                 HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1, &out.bufferSize));
  cudaErrCheck(hipMalloc(&out.dBuffer, out.bufferSize));
  cusparseErrCheck(hipsparseCsr2cscEx2(spHandle, this->h_rows, this->h_columns, this->h_nnz, this->d_csr_values, this->d_csr_offsets,
                                      this->d_csr_columns, out.d_csr_values, out.d_csr_offsets, out.d_csr_columns, HIP_R_64F,
                                      HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1, out.dBuffer));

  cusparseErrCheck(hipsparseCreateCsr(&out.d_matDescr_sparse, out.h_rows, out.h_columns, out.h_nnz, out.d_csr_offsets, out.d_csr_columns,
                                     out.d_csr_values, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));

  // cusparseErrCheck(hipsparseCsrSetPointers(d_matDescr_sparse, d_csr_offsets, d_csr_columns, d_csr_values));

  return out;
};
double Matrix_cuSPARSE::Dnrm2() {
  double h_out;
  int incre = 1;
  cublasErrCheck(hipblasDnrm2(handle, (int)this->h_nnz, this->d_csr_values, incre, &h_out));
  assert(!(h_out != h_out));
  return h_out;
};