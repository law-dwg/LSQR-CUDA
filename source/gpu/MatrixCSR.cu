#include "hip/hip_runtime.h"
#include "MatrixCSR.cuh"
#include <vector>
#define TILE_DIM_X 16
#define TILE_DIM_Y 16

__global__ void spmvNaive(unsigned *rows, unsigned *col, int *rowPtr, int *colIdx, double *val, double *rhs, double *out) {
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  int bid;

  if (gid < *rows) {
    out[gid] = double(0.0);
    for (bid = rowPtr[gid]; bid < rowPtr[gid + 1]; ++bid) {
      out[gid] += val[bid] * rhs[colIdx[bid]];
    }
  }
}

Vector_GPU MatrixCSR::operator*(Vector_GPU &v) { // Multiplication
  Vector_GPU out(this->h_rows, 1);
  unsigned int blocksX = ((this->h_rows * this->h_columns) / (TILE_DIM_X * TILE_DIM_X)) + 1;
  dim3 grid(blocksX, 1, 1);
  dim3 block(TILE_DIM_X * TILE_DIM_X, 1, 1);
  // printf("grid(%d,%d,%d), block(%d,%d,%d)\n", grid.x, grid.y, grid.z, block.x, block.y, block.z);
  spmvNaive<<<grid, block>>>(this->d_rows, this->d_columns, this->d_csrRowPtr, this->d_csrColInd, this->d_csrVal, v.d_mat, out.d_mat);
  return out;
}

MatrixCSR MatrixCSR::transpose() {
  MatrixCSR out(this->h_columns, this->h_rows, this->h_nnz);
  cudaErrCheck(hipMemset(out.d_csrColInd, ZERO, out.h_nnz * sizeof(int)));
  cudaErrCheck(hipMemset(out.d_csrVal, ZERO, out.h_nnz * sizeof(double)));
  cusparseErrCheck(hipsparseCsr2cscEx2_bufferSize(spHandle, this->h_rows, this->h_columns, this->h_nnz, this->d_csrVal, this->d_csrRowPtr,
                                                 this->d_csrColInd, out.d_csrVal, out.d_csrRowPtr, out.d_csrColInd, HIP_R_64F,
                                                 HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1, &out.bufferSize));
  cudaErrCheck(hipMalloc(&out.dBuffer, out.bufferSize));
  cusparseErrCheck(hipsparseCsr2cscEx2(spHandle, this->h_rows, this->h_columns, this->h_nnz, this->d_csrVal, this->d_csrRowPtr, this->d_csrColInd,
                                      out.d_csrVal, out.d_csrRowPtr, out.d_csrColInd, HIP_R_64F, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO,
                                      HIPSPARSE_CSR2CSC_ALG1, out.dBuffer));
  return out;
};