#include "hip/hip_runtime.h"
// CPU
#include "../cpu/lsqr.hpp"
#include "../cpu/utils.hpp"
#include "../cpu/vectorCPU.hpp"
// GPU
#include "matrixCUDA.cuh"
#include "matrixCUSPARSE.cuh"
#include "utils.cuh"
#include "vectorCUBLAS.cuh"
#include "vectorCUDA.cuh"
// Libs
#include <cassert>
#include <chrono>
#include <ctime>
#include <ctype.h>
#include <filesystem>
#include <fstream>
#include <iostream>
#include <math.h>
#include <set>
#include <sstream>
#include <string>
#include <time.h>

template <typename Mat, typename Vec>
float lsqrGPU(unsigned A_r, unsigned A_c, double *A, unsigned b_r, unsigned b_c, double *b, std::string implem, std::string fout) {
  printf("---------------------------------------------\n");
  printf("%s\n\nAx=b where A(%d,%d) and b(%d,1)\n", implem.c_str(), A_r, A_c, b_r);
  cublasStart();
  cusparseStart();
  Mat A_g(A_r, A_c, A);
  Vec b_g(b_r, b_c, b);
  hipEvent_t start, stop;
  cudaErrCheck(hipEventCreate(&start));
  cudaErrCheck(hipEventCreate(&stop));
  cudaErrCheck(hipEventRecord(start));
  Vec x_g = lsqr<Mat, Vec>(A_g, b_g);
  cudaErrCheck(hipDeviceSynchronize());
  cudaErrCheck(hipEventRecord(stop));
  cudaErrCheck(hipEventSynchronize(stop));
  float duration = 0;
  cudaErrCheck(hipEventElapsedTime(&duration, start, stop));
  printf("GPU time used = %f ms for lsqr\n", duration);
  std::string file_out = "output/" + std::to_string(A_c) + "_1_x_" + fout + ".vec";
  VectorCPU x_g_out = x_g.matDeviceToHost();
  writeArrayToFile(file_out, x_g_out.getRows(), x_g_out.getColumns(), x_g_out.getMat());
  cusparseStop();
  cublasStop();
  printf("---------------------------------------------\n");
  return duration;
};

namespace fs = std::filesystem;

int main() {
  double sp;
  std::vector<std::string> fileOut = {"Cpp-DENSE", "CUDA-DENSE", "CUDA-SPARSE", "CUBLAS-DENSE", "CUSPARSE-SPARSE"};
  std::vector<std::string> implementations = {"1. CPU C++ - Dense Matrix Implementation", "2. GPU CUDA - Dense Matrix Implementation",
                                              "3. GPU CUDA - Sparse Matrix Implementation", "4. GPU CUBLAS -  Dense Matrix Implementation",
                                              "5. GPU CUBLAS/CUSPARSE - Sparse Matrix Implementation"};
  printf("\nWelcome to law-dwg's LSQR-CUDA solver!\n\n"
         "The LSQR-algorithm is used to iteratively solve for the vector x in the following expression: "
         "A * x = b,\n"
         "where A is typically a large matrix of size (m*n) and sparsity sp, and b and x are vectors of sizes (m*1) and (n*1), respectively\n\n"
         "LSQR was designed and first authored by C. C. Paige and M. A. Saunders here: "
         "https://web.stanford.edu/group/SOL/software/lsqr/lsqr-toms82a.pdf\n\n"
         "This program runs 5 different implementations of the LSQR algorithm using C++ and CUDA for both dense and sparse A-matricies:\n"
         "    %s\n"
         "    %s\n"
         "    %s\n"
         "    %s\n"
         "    %s\n\n",
         implementations[0].c_str(), implementations[1].c_str(), implementations[2].c_str(), implementations[3].c_str(), implementations[4].c_str());
  printf("This solver requires inputs:\n"
         "    A - 'input/m_n_A_sp.mat'\n"
         "    b - 'input/m_1_b.vec'\n"
         "and writes the solution to:\n"
         "    x - 'output/n_1_x_implementation.vec'\n"
         "    report - 'output/speedups.csv'\n\n");
  std::cout << "Would you like me to make some test data for you? (y/n): ";
  bool matBuild = yesNo();

  if (matBuild) { // build matrices
    unsigned start = 100;
    unsigned end = 100;
    unsigned increment = 100;
    unsigned numOfTests = ((end - start) / increment) + 1;
    printf("\nGreat, I will create %d sets of inputs for you\n\nWhat sparsity should matrix A have? Please enter a number between 0.0-0.95: ",
           numOfTests);
    sp = valInput<double>(0.0, 0.95);
    std::cout << "Building A Matrices of sparsity " << sp << "\n";
    for (int i = start; i <= end; i += increment) {
      matrixBuilder(i, i, sp, "input/", "A");
      matrixBuilder(i, 1, 0, "input/", "b");
    }
  }

  std::string path_name = "input/";
  std::set<fs::path> sorted_by_name;
  for (auto &entry : fs::directory_iterator(path_name)) // alphabetical listing of files in input
    sorted_by_name.insert(entry.path());

  if (sorted_by_name.size() == 0) { // empty input folder
    std::cout << "Looks like there are no files in the input folder. Please add your own matricies in \"NumOfRows_NumOfCols_A.mat\" and "
                 "\"NumOfRows_1_b.vec\" format, or rerun the "
                 "program to autobuild matrices\n"
              << std::endl;
    return 0;
  };
  checkDevice();
  std::set<fs::path>::iterator it = sorted_by_name.begin();

  auto now = std::chrono::system_clock::now();
  std::time_t now_time = std::chrono::system_clock::to_time_t(now);
  tm t = *localtime(&now_time);
  std::cout << std::ctime(&now_time) << std::endl;
  std::ofstream report;
  std::stringstream reportName;
  reportName << "output/" << t.tm_year + 1900 << "-" << t.tm_mon + 1 << "-" << t.tm_mday << "T" << t.tm_hour << ":" << t.tm_min << "_LSQR-CUDA.csv";
  report.open(reportName.str());
  report << "IMPLEMENTATION,A_ROWS,A_COLUMNS,TIME(ms),SPEEDUP\n";
  report.close();
  report.open(reportName.str(), std::ios_base::app);
  while (it != sorted_by_name.end()) { // iterate through sorted files
    cudaErrCheck(hipDeviceReset());
    std::string file1, file2;
    file1 = *it;
    ++it;
    file2 = *it;
    ++it; // iterate every two files
    std::vector<std::string> files{file1, file2};
    unsigned A_rows, A_cols, b_rows, b_cols;
    std::vector<double> A, b;
    for (auto file : files) {
      fileParserLoader(file, A_rows, A_cols, A, b_rows, b_cols, b);
    }
    bool A_sizecheck, b_sizecheck, Ab_rowscheck, b_colscheck, all_checks;
    A_sizecheck = A.size() == A_rows * A_cols && A_rows != 0 && A_cols != 0;
    b_sizecheck = b.size() == b_rows * b_cols && b_rows != 0 && b_cols == 1;
    Ab_rowscheck = A_rows == b_rows;
    all_checks = A_sizecheck && b_sizecheck && Ab_rowscheck;
    if (!all_checks) {
      printf("\n\nERROR, please check the matrix file naming convention (\"NumOfRows_NumOfCols_A_sp.mat\" and "
             "\"NumOfRows_1_b.vec\" format) and make sure the naming convention (rows * columns) matches the number of values in each file\n\n");
      exit(1);
    }
    /** 1. CPU C++ - Dense Matrix Implementation */
    long double CPU_Cpp_ms;
    if (true) {
      printf("---------------------------------------------\n");
      printf("1. CPU C++ - Dense Matrix Implementation\n\nAx=b where A(%d,%d) and b(%d,1)\n", A_rows, A_cols, b_rows);
      VectorCPU A_c(A_rows, A_cols, A.data());
      VectorCPU b_c(b_rows, b_cols, b.data());
      std::clock_t c_start = std::clock();
      VectorCPU x_c = lsqr<VectorCPU, VectorCPU>(A_c, b_c);
      std::clock_t c_end = std::clock();
      CPU_Cpp_ms = 1000.0 * (c_end - c_start) / CLOCKS_PER_SEC;
      std::cout << "CPU time used = " << CPU_Cpp_ms << " ms for lsqr\n";
      std::string file_out = "output/" + std::to_string(A_cols) + "_1_x_" + fileOut[0] + ".vec";
      writeArrayToFile(file_out, x_c.getRows(), x_c.getColumns(), x_c.getMat());
      printf("---------------------------------------------\n");
    }
    report << fileOut[0] << "," << std::to_string(A_rows) << "," << std::to_string(A_cols) << "," << std::to_string(CPU_Cpp_ms) << ","
           << "0"
           << "\n";
    /** 2. GPU CUDA - Dense Matrix Implementation */
    float CUDA_DENSE_ms = lsqrGPU<VectorCUDA, VectorCUDA>(A_rows, A_cols, A.data(), b_rows, b_cols, b.data(), implementations[1], fileOut[1]);
    cudaErrCheck(hipDeviceReset());
    report << fileOut[1] << "," << std::to_string(A_rows) << "," << std::to_string(A_cols) << "," << std::to_string(CUDA_DENSE_ms) << ","<< CPU_Cpp_ms/CUDA_DENSE_ms <<"\n";;

    float CUDA_SPARSE_ms = lsqrGPU<MatrixCUDA, VectorCUDA>(A_rows, A_cols, A.data(), b_rows, b_cols, b.data(), implementations[2], fileOut[2]);
    cudaErrCheck(hipDeviceReset());
    report << fileOut[2] << "," << std::to_string(A_rows) << "," << std::to_string(A_cols) << "," << std::to_string(CUDA_SPARSE_ms) << ","<< CPU_Cpp_ms/CUDA_SPARSE_ms <<"\n";
    
    float CUBLAS_DENSE_ms = lsqrGPU<VectorCUBLAS, VectorCUBLAS>(A_rows, A_cols, A.data(), b_rows, b_cols, b.data(), implementations[3], fileOut[3]);
    cudaErrCheck(hipDeviceReset());
    report << fileOut[3] << "," << std::to_string(A_rows) << "," << std::to_string(A_cols) << "," << std::to_string(CUBLAS_DENSE_ms) << ","<< CPU_Cpp_ms/CUBLAS_DENSE_ms <<"\n";
    
    float CUSPARSE_SPARSE_ms =
        lsqrGPU<MatrixCUSPARSE, VectorCUBLAS>(A_rows, A_cols, A.data(), b_rows, b_cols, b.data(), implementations[4], fileOut[4]);
    cudaErrCheck(hipDeviceReset());
    report << fileOut[4] << "," << std::to_string(A_rows) << "," << std::to_string(A_cols) << "," << std::to_string(CUSPARSE_SPARSE_ms) << "," << CPU_Cpp_ms/CUSPARSE_SPARSE_ms <<"\n";
    
    cudaLastErrCheck();
  }
  report.close();
}