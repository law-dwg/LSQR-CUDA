#include "MatrixCUSPARSE.cuh"
#include <assert.h>
Vector_GPU MatrixCUSPARSE::operator*(Vector_GPU &v) {
  Vector_GPU out(this->h_rows, 1);
  hipsparseDnVecDescr_t rhs_desc, out_desc;
  cusparseErrCheck(hipsparseCreateDnVec(&rhs_desc, out.getRows(), v.d_mat, HIP_R_64F));
  cusparseErrCheck(hipsparseCreateDnVec(&out_desc, out.getRows(), out.d_mat, HIP_R_64F));
  cusparseErrCheck(hipsparseSpMV_bufferSize(spHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &ONE, this->d_matDescr_sparse, rhs_desc, &ZERO, out_desc,
                                           HIP_R_64F, HIPSPARSE_MV_ALG_DEFAULT, &bufferSize));
  // cudaErrCheck(hipMalloc(&dBuffer, this->bufferSize));
  cusparseErrCheck(hipsparseSpMV(spHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &ONE, this->d_matDescr_sparse, rhs_desc, &ZERO, out_desc, HIP_R_64F,
                                HIPSPARSE_MV_ALG_DEFAULT, dBuffer));

  return out;
};
Vector_CUBLAS MatrixCUSPARSE::operator*(Vector_CUBLAS &v) {
  Vector_CUBLAS out(this->h_rows, 1);
  hipsparseDnVecDescr_t rhs_desc, out_desc;
  cusparseErrCheck(hipsparseCreateDnVec(&rhs_desc, out.getRows(), v.d_mat, HIP_R_64F));
  cusparseErrCheck(hipsparseCreateDnVec(&out_desc, out.getRows(), out.d_mat, HIP_R_64F));
  cusparseErrCheck(hipsparseSpMV_bufferSize(spHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &ONE, this->d_matDescr_sparse, rhs_desc, &ZERO, out_desc,
                                           HIP_R_64F, HIPSPARSE_MV_ALG_DEFAULT, &bufferSize));
  // cudaErrCheck(hipMalloc(&dBuffer, this->bufferSize));
  cusparseErrCheck(hipsparseSpMV(spHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &ONE, this->d_matDescr_sparse, rhs_desc, &ZERO, out_desc, HIP_R_64F,
                                HIPSPARSE_MV_ALG_DEFAULT, dBuffer));

  return out;
};
MatrixCUSPARSE MatrixCUSPARSE::transpose() {
  MatrixCUSPARSE out(this->h_columns, this->h_rows, this->h_nnz);
  cudaErrCheck(hipMemset(out.d_csr_columns, ZERO, out.h_nnz * sizeof(int)));
  cudaErrCheck(hipMemset(out.d_csr_values, ZERO, out.h_nnz * sizeof(double)));
  cusparseErrCheck(hipsparseCsr2cscEx2_bufferSize(spHandle, this->h_rows, this->h_columns, this->h_nnz, this->d_csr_values, this->d_csr_offsets,
                                                 this->d_csr_columns, out.d_csr_values, out.d_csr_offsets, out.d_csr_columns, HIP_R_64F,
                                                 HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1, &out.bufferSize));
  cudaErrCheck(hipMalloc(&out.dBuffer, out.bufferSize));
  cusparseErrCheck(hipsparseCsr2cscEx2(spHandle, this->h_rows, this->h_columns, this->h_nnz, this->d_csr_values, this->d_csr_offsets,
                                      this->d_csr_columns, out.d_csr_values, out.d_csr_offsets, out.d_csr_columns, HIP_R_64F,
                                      HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1, out.dBuffer));

  cusparseErrCheck(hipsparseCreateCsr(&out.d_matDescr_sparse, out.h_rows, out.h_columns, out.h_nnz, out.d_csr_offsets, out.d_csr_columns,
                                     out.d_csr_values, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));
  return out;
};
double MatrixCUSPARSE::Dnrm2() {
  double h_out;
  int incre = 1;
  cublasErrCheck(hipblasDnrm2(handle, (int)this->h_nnz, this->d_csr_values, incre, &h_out));
  assert(!(h_out != h_out));
  return h_out;
};