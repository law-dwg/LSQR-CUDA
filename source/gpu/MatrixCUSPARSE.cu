#include "MatrixCUSPARSE.cuh"
#include <assert.h>
Vector_GPU MatrixCUSPARSE::operator*(Vector_GPU &v) {
  Vector_GPU out(this->h_rows, 1);
  hipsparseDnVecDescr_t rhs_desc, out_desc;
  cusparseErrCheck(hipsparseCreateDnVec(&rhs_desc, out.getRows(), v.d_mat, HIP_R_64F));
  cusparseErrCheck(hipsparseCreateDnVec(&out_desc, out.getRows(), out.d_mat, HIP_R_64F));
  cusparseErrCheck(hipsparseSpMV_bufferSize(spHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &ONE, this->spMatDescr, rhs_desc, &ZERO, out_desc, HIP_R_64F,
                                           HIPSPARSE_MV_ALG_DEFAULT, &bufferSize));
  // cudaErrCheck(hipMalloc(&dBuffer, this->bufferSize));
  cusparseErrCheck(hipsparseSpMV(spHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &ONE, this->spMatDescr, rhs_desc, &ZERO, out_desc, HIP_R_64F,
                                HIPSPARSE_MV_ALG_DEFAULT, dBuffer));

  return out;
};
Vector_CUBLAS MatrixCUSPARSE::operator*(Vector_CUBLAS &v) {
  Vector_CUBLAS out(this->h_rows, 1);
  hipsparseDnVecDescr_t rhs_desc, out_desc;
  cusparseErrCheck(hipsparseCreateDnVec(&rhs_desc, out.getRows(), v.d_mat, HIP_R_64F));
  cusparseErrCheck(hipsparseCreateDnVec(&out_desc, out.getRows(), out.d_mat, HIP_R_64F));
  cusparseErrCheck(hipsparseSpMV_bufferSize(spHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &ONE, this->spMatDescr, rhs_desc, &ZERO, out_desc, HIP_R_64F,
                                           HIPSPARSE_MV_ALG_DEFAULT, &bufferSize));
  // cudaErrCheck(hipMalloc(&dBuffer, this->bufferSize));
  cusparseErrCheck(hipsparseSpMV(spHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &ONE, this->spMatDescr, rhs_desc, &ZERO, out_desc, HIP_R_64F,
                                HIPSPARSE_MV_ALG_DEFAULT, dBuffer));

  return out;
};
MatrixCUSPARSE MatrixCUSPARSE::transpose() {
  MatrixCUSPARSE out(this->h_columns, this->h_rows, this->h_nnz);
  cudaErrCheck(hipMemset(out.d_csrColInd, ZERO, out.h_nnz * sizeof(int)));
  cudaErrCheck(hipMemset(out.d_csrVal, ZERO, out.h_nnz * sizeof(double)));
  cusparseErrCheck(hipsparseCsr2cscEx2_bufferSize(spHandle, this->h_rows, this->h_columns, this->h_nnz, this->d_csrVal, this->d_csrRowPtr,
                                                 this->d_csrColInd, out.d_csrVal, out.d_csrRowPtr, out.d_csrColInd, HIP_R_64F,
                                                 HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1, &out.bufferSize));
  cudaErrCheck(hipMalloc(&out.dBuffer, out.bufferSize));
  cusparseErrCheck(hipsparseCsr2cscEx2(spHandle, this->h_rows, this->h_columns, this->h_nnz, this->d_csrVal, this->d_csrRowPtr, this->d_csrColInd,
                                      out.d_csrVal, out.d_csrRowPtr, out.d_csrColInd, HIP_R_64F, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO,
                                      HIPSPARSE_CSR2CSC_ALG1, out.dBuffer));

  cusparseErrCheck(hipsparseCreateCsr(&out.spMatDescr, out.h_rows, out.h_columns, out.h_nnz, out.d_csrRowPtr, out.d_csrColInd, out.d_csrVal,
                                     HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));
  return out;
};
double MatrixCUSPARSE::Dnrm2() {
  double h_out;
  int incre = 1;
  cublasErrCheck(hipblasDnrm2(handle, (int)this->h_nnz, this->d_csrVal, incre, &h_out));
  assert(!(h_out != h_out));
  return h_out;
};