#include "hip/hip_runtime.h"
#include "MatrixCUDA.cuh"
#include "VectorCUDA.cuh"
#include <assert.h>
#include <vector>
#define TILE_DIM_X 16
#define TILE_DIM_Y 16

__global__ void spmvNaive(unsigned *rows, unsigned *col, int *rowPtr, int *colIdx, double *val, double *rhs, double *out) {
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  int bid;

  if (gid < *rows) {
    out[gid] = double(0.0);
    for (bid = rowPtr[gid]; bid < rowPtr[gid + 1]; ++bid) {
      out[gid] += val[bid] * rhs[colIdx[bid]];
    }
  }
}

VectorCUDA MatrixCUDA::operator*(VectorCUDA &v) { // Multiplication
  VectorCUDA out(this->h_rows, 1);
  unsigned blocksX = ((this->h_rows * this->h_columns) / (TILE_DIM_X * TILE_DIM_X)) + 1;
  dim3 grid(blocksX, 1, 1);
  dim3 block(TILE_DIM_X * TILE_DIM_X, 1, 1);
  // printf("grid(%d,%d,%d), block(%d,%d,%d)\n", grid.x, grid.y, grid.z, block.x, block.y, block.z);
  spmvNaive<<<grid, block>>>(this->d_rows, this->d_columns, this->d_csrRowPtr, this->d_csrColInd, this->d_csrVal, v.d_mat, out.d_mat);
  return out;
}

MatrixCUDA MatrixCUDA::transpose() {
  MatrixCUDA out(this->h_columns, this->h_rows, this->h_nnz);
  cudaErrCheck(hipMemset(out.d_csrColInd, ZERO, out.h_nnz * sizeof(int)));
  cudaErrCheck(hipMemset(out.d_csrVal, ZERO, out.h_nnz * sizeof(double)));
  cusparseErrCheck(hipsparseCsr2cscEx2_bufferSize(spHandle, this->h_rows, this->h_columns, this->h_nnz, this->d_csrVal, this->d_csrRowPtr,
                                                 this->d_csrColInd, out.d_csrVal, out.d_csrRowPtr, out.d_csrColInd, HIP_R_64F,
                                                 HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1, &out.bufferSize));
  cudaErrCheck(hipMalloc(&out.dBuffer, out.bufferSize));
  cusparseErrCheck(hipsparseCsr2cscEx2(spHandle, this->h_rows, this->h_columns, this->h_nnz, this->d_csrVal, this->d_csrRowPtr, this->d_csrColInd,
                                      out.d_csrVal, out.d_csrRowPtr, out.d_csrColInd, HIP_R_64F, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO,
                                      HIPSPARSE_CSR2CSC_ALG1, out.dBuffer));
  return out;
};

double MatrixCUDA::Dnrm2() {
  dim3 threads(TILE_DIM_X, 1);
  int blockX = ((this->h_nnz + TILE_DIM_X - 1) / TILE_DIM_X);
  dim3 blocks(blockX, 1);
  double *d_out, *d_max;
  unsigned *d_ONE;
  double zero = 0.0;
  unsigned one = 1;
  double h_max;
  double h_out;
  cudaErrCheck(hipMalloc(&d_out, sizeof(double)));
  cudaErrCheck(hipMalloc(&d_max, sizeof(double)));
  cudaErrCheck(hipMalloc(&d_ONE, sizeof(unsigned)));
  cudaErrCheck(hipMemcpy(d_out, &zero, sizeof(double), hipMemcpyHostToDevice));
  cudaErrCheck(hipMemcpy(d_max, &zero, sizeof(double), hipMemcpyHostToDevice));
  cudaErrCheck(hipMemcpy(d_ONE, &one, sizeof(unsigned), hipMemcpyHostToDevice));

  // printf("dnrm2 threads(%d x %d)=%d, blocks(%d, %d)=%d\n", threads.x, threads.y, threads.x * threads.y, blocks.x, blocks.y, blocks.x * blocks.y);
  // unsigned s_mem = sizeof(double) * TILE_DIM_X;
  printf("h_nnz=%d\n", this->h_nnz);
  maxVal<<<blocks, threads, 16 * sizeof(double)>>>(this->d_csrVal, this->d_nnz, d_ONE, d_max);
  // cudaErrCheck(hipPeekAtLastError());
  cudaErrCheck(hipDeviceSynchronize());
  // unsigned s_mem = sizeof(double) * TILE_DIM_X;
  dnrm2<<<blocks, threads, 16 * sizeof(double)>>>(this->d_csrVal, this->d_nnz, d_ONE, d_max, d_out);
  cudaErrCheck(hipDeviceSynchronize());
  cudaErrCheck(hipMemcpy(&h_out, d_out, sizeof(double), hipMemcpyDeviceToHost));
  cudaErrCheck(hipMemcpy(&h_max, d_max, sizeof(double), hipMemcpyDeviceToHost));
  assert(!(h_out != h_out));
  assert(h_out > 0);
  return (std::abs(h_max) * sqrt(h_out));
}