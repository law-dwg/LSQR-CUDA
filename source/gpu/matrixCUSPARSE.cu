#include "matrixCUSPARSE.cuh"
#include <assert.h>

template <typename T> void SpMV(MatrixCUSPARSE &M, T &v, T &out) {
  if (M.getColumns() == v.getRows()) {
    hipsparseDnVecDescr_t rhs_desc, out_desc;
    cusparseErrCheck(hipsparseCreateDnVec(&rhs_desc, v.getRows(), v.getMat(), HIP_R_64F));
    cusparseErrCheck(hipsparseCreateDnVec(&out_desc, out.getRows(), out.getMat(), HIP_R_64F));
    cusparseErrCheck(hipsparseSpMV_bufferSize(spHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &ONE, M.spMatDescr, rhs_desc, &ZERO, out_desc, HIP_R_64F,
                                             HIPSPARSE_MV_ALG_DEFAULT, &M.bufferSize));
    // cudaErrCheck(hipMalloc(&dBuffer, this->bufferSize));
    cusparseErrCheck(hipsparseSpMV(spHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &ONE, M.spMatDescr, rhs_desc, &ZERO, out_desc, HIP_R_64F,
                                  HIPSPARSE_MV_ALG_DEFAULT, M.dBuffer));
  } else {
    printf("Cannot perform multiplication, dimension mismatch %s(%d)\n", __FILE__, __LINE__);
    exit(1);
  }
};

VectorCUDA MatrixCUSPARSE::operator*(VectorCUDA &v) {
  VectorCUDA out(this->h_rows, 1);
  SpMV(*this, v, out);
  return out;
};

VectorCUBLAS MatrixCUSPARSE::operator*(VectorCUBLAS &v) {
  VectorCUBLAS out(this->h_rows, 1);
  SpMV(*this, v, out);
  return out;
};

MatrixCUSPARSE MatrixCUSPARSE::transpose() {
  MatrixCUSPARSE out(this->h_columns, this->h_rows, this->h_nnz);
  cudaErrCheck(hipMemset(out.d_csrColInd, ZERO, out.h_nnz * sizeof(int)));
  cudaErrCheck(hipMemset(out.d_csrVal, ZERO, out.h_nnz * sizeof(double)));
  cusparseErrCheck(hipsparseCsr2cscEx2_bufferSize(spHandle, this->h_rows, this->h_columns, this->h_nnz, this->d_csrVal, this->d_csrRowPtr,
                                                 this->d_csrColInd, out.d_csrVal, out.d_csrRowPtr, out.d_csrColInd, HIP_R_64F,
                                                 HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1, &out.bufferSize));
  cudaErrCheck(hipMalloc(&out.dBuffer, out.bufferSize));
  cusparseErrCheck(hipsparseCsr2cscEx2(spHandle, this->h_rows, this->h_columns, this->h_nnz, this->d_csrVal, this->d_csrRowPtr, this->d_csrColInd,
                                      out.d_csrVal, out.d_csrRowPtr, out.d_csrColInd, HIP_R_64F, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO,
                                      HIPSPARSE_CSR2CSC_ALG1, out.dBuffer));

  cusparseErrCheck(hipsparseCreateCsr(&out.spMatDescr, out.h_rows, out.h_columns, out.h_nnz, out.d_csrRowPtr, out.d_csrColInd, out.d_csrVal,
                                     HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));
  return out;
};

double MatrixCUSPARSE::Dnrm2() {
  double h_out;
  int incre = 1;
  cublasErrCheck(hipblasDnrm2(handle, (int)this->h_nnz, this->d_csrVal, incre, &h_out));
  // sanity checks
  assert(!(h_out != h_out));
  assert(0 < h_out);
  return h_out;
};