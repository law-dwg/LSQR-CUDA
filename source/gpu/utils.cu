#include "hipblas.h"
#include "utils.cuh"
#include <hip/hip_runtime.h>

hipError_t cudaStat;
hipblasHandle_t handle;
hipStream_t stream;
hipblasStatus_t stat1 = hipblasCreate(&handle);
hipError_t cudaStat1 = hipStreamCreate(&stream);
hipblasStatus_t stat = hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);
hipblasStatus_t stat2 = hipblasSetStream(handle, stream);
void cublasReset() {
  stat1 = hipblasCreate(&handle);
  cudaStat1 = hipStreamCreate(&stream);
  stat = hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);
  stat2 = hipblasSetStream(handle, stream);
}