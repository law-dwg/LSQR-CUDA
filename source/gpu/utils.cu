#include "hipblas.h"
#include "utils.cuh"
#include <hip/hip_runtime.h>
#include <iostream>
#define MODE HIPBLAS_POINTER_MODE_HOST

hipblasStatus_t stat;
hipError_t cudaStat;
hipblasHandle_t handle;
hipStream_t stream;
hipblasStatus_t statCreateHandle;
hipError_t cudaStatCreateStream;
hipblasStatus_t statSetStream;
hipblasStatus_t statSetPointerMode;

void cublasStart() {
  statCreateHandle = hipblasCreate(&handle);
  std::cout << cublasGetErrorString(statCreateHandle) << std::endl;
  if (statCreateHandle != HIPBLAS_STATUS_SUCCESS) {
    printf("CUBLAS create handle failed\n");
    // return EXIT_FAILURE;
  }
  cudaStatCreateStream = hipStreamCreate(&stream);
  if (cudaStatCreateStream != hipSuccess) {
    printf("CUBLAS create stream failed\n");
    // return EXIT_FAILURE;
  }
  statSetStream = hipblasSetStream(handle, stream);
  if (statSetStream != HIPBLAS_STATUS_SUCCESS) {
    printf("CUBLAS set pointer failed\n");
    // return EXIT_FAILURE;
  }
  statSetPointerMode = hipblasSetPointerMode(handle, MODE);
  if (statSetPointerMode != HIPBLAS_STATUS_SUCCESS) {
    printf("CUBLAS set pointer failed\n");
    // return EXIT_FAILURE;
  }
};
void cublasStop() { hipblasDestroy(handle); };
int checkDevice() {
  // Check Cuda Capabale Device
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  int device;
  if (deviceCount > 0) {
    for (device = 0; device < deviceCount; ++device) {
      hipDeviceProp_t deviceProp;
      hipGetDeviceProperties(&deviceProp, device);
      printf("Device %s has compute capability %d.%d.\n", deviceProp.name, deviceProp.major, deviceProp.minor);
      printf("Number of multiprocessors: %d\n", deviceProp.multiProcessorCount);
      printf("Clock rate: %d Hz\n", deviceProp.clockRate);
      printf("Total amount of global memory: %d KB\n", deviceProp.totalGlobalMem / 1024);
      printf("Total amount of constant memory: %d KB\n", deviceProp.totalConstMem / 1024);
      printf("Total amount of shared memory per block: %d KB\n", deviceProp.sharedMemPerBlock / 1024);
      printf("Total amount of shared memory per SM: %d KB\n", 64);
      printf("Warp size: %d\n", deviceProp.warpSize);
      printf("Maximum number of threads per block: %d\n", deviceProp.maxThreadsPerBlock);
      printf("Maximum number of blocks per multiprocessor: %d\n", deviceProp.maxThreadsPerMultiProcessor / deviceProp.maxThreadsPerBlock);
      printf("Maximum number of threads per multiprocessor: %d\n", deviceProp.maxThreadsPerMultiProcessor);
      printf("Maximum number of warps per multiprocessor: %d\n", deviceProp.maxThreadsPerMultiProcessor / 32);
      printf("Maximum Grid size: (%d,%d,%d)\n", deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
      printf("Maximum block dimension: (%d,%d,%d)\n", deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
    }
  } else {
    printf("NO CUDA DEVICE AVAILABLE");
  }
  return deviceCount;
};
const char *cublasGetErrorString(hipblasStatus_t status) {
  switch (status) {
  case HIPBLAS_STATUS_SUCCESS:
    return "HIPBLAS_STATUS_SUCCESS";
  case HIPBLAS_STATUS_NOT_INITIALIZED:
    return "HIPBLAS_STATUS_NOT_INITIALIZED";
  case HIPBLAS_STATUS_ALLOC_FAILED:
    return "HIPBLAS_STATUS_ALLOC_FAILED";
  case HIPBLAS_STATUS_INVALID_VALUE:
    return "HIPBLAS_STATUS_INVALID_VALUE";
  case HIPBLAS_STATUS_ARCH_MISMATCH:
    return "HIPBLAS_STATUS_ARCH_MISMATCH";
  case HIPBLAS_STATUS_MAPPING_ERROR:
    return "HIPBLAS_STATUS_MAPPING_ERROR";
  case HIPBLAS_STATUS_EXECUTION_FAILED:
    return "HIPBLAS_STATUS_EXECUTION_FAILED";
  case HIPBLAS_STATUS_INTERNAL_ERROR:
    return "HIPBLAS_STATUS_INTERNAL_ERROR";
  }
  return "unknown error";
};