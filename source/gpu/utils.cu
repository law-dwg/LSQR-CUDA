#include "hipblas.h"
#include "utils.cuh"
#include <hip/hip_runtime.h>
#include <iostream>
#define MODE HIPBLAS_POINTER_MODE_HOST

hipblasStatus_t stat;
hipError_t cudaStat;
hipblasHandle_t handle;
hipStream_t stream;
hipblasStatus_t statCreateHandle = hipblasCreate(&handle);
hipError_t cudaStatCreateStream = hipStreamCreate(&stream);
hipblasStatus_t statSetStream = hipblasSetStream(handle, stream);
hipblasStatus_t statSetPointerMode = hipblasSetPointerMode(handle, MODE);

void cublasReset() {
  statCreateHandle = hipblasCreate(&handle);
  std::cout << cublasGetErrorString(statCreateHandle) << std::endl;
  if (statCreateHandle != HIPBLAS_STATUS_SUCCESS) {
    printf("CUBLAS create handle failed\n");
    // return EXIT_FAILURE;
  }
  cudaStatCreateStream = hipStreamCreate(&stream);
  if (cudaStatCreateStream != hipSuccess) {
    printf("CUBLAS create stream failed\n");
    // return EXIT_FAILURE;
  }
  statSetStream = hipblasSetStream(handle, stream);
  if (statSetStream != HIPBLAS_STATUS_SUCCESS) {
    printf("CUBLAS set pointer failed\n");
    // return EXIT_FAILURE;
  }
  statSetPointerMode = hipblasSetPointerMode(handle, MODE);
  if (statSetPointerMode != HIPBLAS_STATUS_SUCCESS) {
    printf("CUBLAS set pointer failed\n");
    // return EXIT_FAILURE;
  }
};
const char *cublasGetErrorString(hipblasStatus_t status) {
  switch (status) {
  case HIPBLAS_STATUS_SUCCESS:
    return "HIPBLAS_STATUS_SUCCESS";
  case HIPBLAS_STATUS_NOT_INITIALIZED:
    return "HIPBLAS_STATUS_NOT_INITIALIZED";
  case HIPBLAS_STATUS_ALLOC_FAILED:
    return "HIPBLAS_STATUS_ALLOC_FAILED";
  case HIPBLAS_STATUS_INVALID_VALUE:
    return "HIPBLAS_STATUS_INVALID_VALUE";
  case HIPBLAS_STATUS_ARCH_MISMATCH:
    return "HIPBLAS_STATUS_ARCH_MISMATCH";
  case HIPBLAS_STATUS_MAPPING_ERROR:
    return "HIPBLAS_STATUS_MAPPING_ERROR";
  case HIPBLAS_STATUS_EXECUTION_FAILED:
    return "HIPBLAS_STATUS_EXECUTION_FAILED";
  case HIPBLAS_STATUS_INTERNAL_ERROR:
    return "HIPBLAS_STATUS_INTERNAL_ERROR";
  }
  return "unknown error";
};