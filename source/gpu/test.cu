//#include "../cpu/lsqr_cpu.h"
#include "../cpu/matVec_cpu.h"
#include ""
#include "lsqr_gpu.cuh"
//#include "matCsr_gpu.cuh"
#include "matVec_gpu.cuh"
#include <assert.h>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <limits>
#include <sstream>
#include <stdio.h>  //NULL, printf
#include <stdlib.h> //srand, rand
#include <string.h>
#include <time.h>

int main() {
  checkDevice();
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  // int a_csr_rows, b_csr_rows, a_csr_cols, b_csr_cols;
  // a_csr_rows = b_csr_rows = a_csr_cols = b_csr_cols = 3;
  // double *a_csr_heap = new double[9]{0, 1, 2, 4, 0, 5, 6, 7, 0};
  // Matrix_CPU S(33, 1);
  // double *b_heap = new double[3]{1, 2, 3};
  // // double *b_heap = new double[100];
  // // for (int i = 0; i < (100); ++i) {
  // //  b_heap[i] = i;
  // //}
  // Matrix_CSR_GPU A(a_csr_rows, a_csr_cols, a_csr_heap);
  // // Matrix_CSR_GPU A(S.rows, S.columns, S.getMat());
  // Vector_GPU b(3, 1, b_heap);
  // A *b;
  // delete a_csr_heap, b_heap;

  double *a_heap, *b_heap;
  int a_rows, b_rows, a_cols, b_cols;
  a_rows = b_rows = 100;
  a_cols = 100;
  b_cols = 1;
  a_heap = new double[a_rows * a_cols];
  b_heap = new double[b_rows * b_cols];
  for (int i = 0; i < (a_rows * a_cols); i++) {
    if (i < b_rows) {
      a_heap[i] = i;
      b_heap[i] = i;
    } else {
      a_heap[i] = i;
    }
  }

  // writeArrayToFile("input/A.txt", a_rows * a_cols, a_heap);
  // writeArrayToFile("input/b.txt", b_rows * b_cols, b_heap);
  Vector_CPU A_c(a_rows, a_cols, a_heap);
  Vector_CPU b_c(b_rows, b_cols, b_heap);
  Vector_GPU A_g(a_rows, a_cols, a_heap);
  Vector_GPU b_g(b_rows, b_cols, b_heap);
  // Vector_GPU C_g = ((b_g.transpose() * b_g) * b_g.Dnrm2()) * A_g.Dnrm2();
  // Vector_CPU C_c =  ((b_c.transpose() * b_c) * b_g.Dnrm2()) * A_c.Dnrm2();
  Vector_GPU C_g = ((A_g + A_g.transpose()) * 3 * A_g) - (A_g * 1.8) * A_g.Dnrm2();
  Vector_CPU C_c = ((A_c + A_c.transpose()) * 3 * A_c) - (A_c * 1.8) * A_c.Dnrm2();
  // Vector_GPU C_g = A_g.transpose()*1.8;
  // Vector_CPU C_c = A_c.transpose()*1.8;
  C_g = C_g.transpose();
  C_c = C_c.transpose();
  Vector_CPU C_g_out = C_g.matDeviceToHost();
  C_g_out.print();
  C_c.print();
  bool ans = compareMat(C_g_out.getMat(), C_g_out.getRows(), C_g_out.getColumns(), C_c.getMat(), C_c.getRows(), C_c.getColumns());
  // Matrix_GPU c_c(b_rows, b_cols, b_heap);
  // Matrix_GPU A = A_c;
  // Vector_GPU b = b_c;
  // printf("STARTING LSQR\n");
  // Vector_GPU x_G = lsqr_gpu(A, b);
  // Vector_CPU x_C = lsqr_cpu(A_c, b_c);
  // Vector_CPU x_G_out = x_G.matDeviceToHost();
  // bool ans = compareMat(x_G_out.getMat(), x_G_out.getRows(), x_G_out.getColumns(), x_C.getMat(), x_C.getRows(), x_C.getColumns());
  // x_G_out.print();
  // x_C.print();
  // writeArrayToFile("output/x_c.txt", x_G_out.getRows() * x_G_out.getColumns(), x_G_out.getMat());
  // writeArrayToFile("output/x_g.txt", x_C.getRows() * x_C.getColumns(), x_C.getMat());
  delete a_heap, b_heap;
  hipError_t err = hipGetLastError(); // add
  if (err != hipSuccess) {
    std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl;
  } // add
  hipProfilerStop();

  return 0;
};