#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "matVec_cublas.cuh"
#include "utils.cuh"
#include <assert.h>
#include <stdio.h>
const double ONE = 1.0;
const double ZERO = 0.0;
const double NEGONE = -1.0;
// void __global__ print(double *input, unsigned *r, unsigned *c) {
//  const unsigned int bid = blockIdx.x                               // 1D
//                           + blockIdx.y * gridDim.x                 // 2D
//                           + gridDim.x * gridDim.y * blockIdx.z;    // 3D
//  const unsigned int threadsPerBlock = blockDim.x * blockDim.y      // 2D
//                                       * blockDim.z;                // 3D
//  const unsigned int tid = threadIdx.x                              // 1D
//                           + threadIdx.y * blockDim.x               // 2D
//                           + blockDim.x * blockDim.x * threadIdx.z; // 3D
//  const unsigned int gid = bid * threadsPerBlock + tid;
//  // printf("thread(%d,%d,%d), block(%d,%d,%d), bid=%d, gid=%d,
//  // value=%f\n",threadIdx.x,threadIdx.y,threadIdx.z,
//  //    blockIdx.x,blockIdx.y,blockIdx.z,bid,gid,input[gid]);
//  __syncthreads();
//  if (gid < *r * *c) {
//    printf("%f\n", input[gid]);
//  }
//}
/** Operator overloads */
Vector_CUBLAS Vector_CUBLAS::operator*(Vector_CUBLAS &v) {
  Vector_CUBLAS out(this->h_rows, v.h_columns);
  int m = out.h_rows;
  int n = out.h_columns;
  int k = this->h_columns;
  int lda = m, ldb = k, ldc = m;
  // printf("m=%d,n=%d,k=%d,lda=%d, ldb=%d, ldc=%d, d_rows=%d, d_cols=%d,\n", m, n, k, lda, ldb, ldc, rows, cols);
  stat = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &ONE, this->d_mat, lda, v.d_mat, ldb, &ZERO, out.d_mat, ldc);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    std::cout << cublasGetErrorString(stat) << std::endl;
    printf("CUBLAS matrix mult failed\n");
  }

  return out;
};
Vector_CUBLAS Vector_CUBLAS::operator*(double i) {
  Vector_CUBLAS out(this->h_rows, this->h_columns);
  stat = hipblasDcopy(handle, this->h_rows * this->h_columns, this->d_mat, 1, out.d_mat, 1);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    std::cout << cublasGetErrorString(stat) << std::endl;
    printf("CUBLAS scale failed\n");
  }
  stat = hipblasDscal(handle, this->h_rows * this->h_columns, &i, out.d_mat, 1);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    std::cout << cublasGetErrorString(stat) << std::endl;
    printf("CUBLAS scale failed\n");
  }
  return out;
};
void Vector_CUBLAS::operator=(Vector_CPU &v){};

Vector_CUBLAS Vector_CUBLAS::operator-(const Vector_CUBLAS &v) {
  Vector_CUBLAS out(this->h_rows, this->h_columns);
  if (this->h_rows * this->h_columns == v.h_rows * v.h_columns) {
    int m = out.h_rows;
    int n = out.h_columns;
    int k = this->h_columns;
    int lda = m, ldb = k, ldc = m;
    hipblasOperation_t OP = (m == 1 || n == 1) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    // printf("h_rows = %d, h_cols = %d, v.h_rows = %d, v.h_cols = %d\n",this->h_rows,this->h_columns,v.h_rows,v.h_columns);
    stat = hipblasDgeam(handle, HIPBLAS_OP_N, OP, m, n, &ONE, this->d_mat, lda, &NEGONE, v.d_mat, ldb, out.d_mat, ldc); // ADD-SUB-TRANSPOSE
    hipDeviceSynchronize();
    if (stat != HIPBLAS_STATUS_SUCCESS) {
      std::cout << cublasGetErrorString(stat) << std::endl;
      printf("CUBLAS matrix subract failed\n");
    }
  } else
    printf("CUBLAS SUBTRACT ERROR, MATRICES ARENT SAME SIZE\n");
  return out;
};
Vector_CUBLAS Vector_CUBLAS::operator+(const Vector_CUBLAS &v) {
  Vector_CUBLAS out(this->h_rows, this->h_columns);
  // printf("h_rows = %d, h_cols = %d, v.h_rows = %d, v.h_cols = %d\n",this->h_rows,this->h_columns,v.h_rows,v.h_columns);
  if (this->h_rows * this->h_columns == v.h_rows * v.h_columns) {
    int m = out.h_rows;
    int n = out.h_columns;
    int k = this->h_columns;
    int lda = m, ldb = k, ldc = m;
    hipblasOperation_t OP = (m == 1 || n == 1) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    stat = hipblasDgeam(handle, HIPBLAS_OP_N, OP, m, n, &ONE, this->d_mat, lda, &ONE, v.d_mat, ldb, out.d_mat, ldc); // ADD-SUB-TRANSPOSE
    hipDeviceSynchronize();
    if (stat != HIPBLAS_STATUS_SUCCESS) {
      std::cout << cublasGetErrorString(stat) << std::endl;
      printf("CUBLAS matrix addition failed\n");
    }
  } else {
    printf("CUBLAS ADDITION ERROR, MATRICES ARENT SAME SIZE\n");
  }
  return out;
};

/** Member functions */
// void printmat();
Vector_CPU Vector_CUBLAS::matDeviceToHost() {
  double *out = new double[this->h_columns * this->h_rows]; // heap to prevent a stack overflow
  unsigned int rows;
  unsigned int cols;
  gpuErrchk(hipMemcpy(out, this->d_mat, sizeof(double) * this->h_columns * this->h_rows, hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(&rows, this->d_rows, sizeof(unsigned int), hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(&cols, this->d_columns, sizeof(unsigned int), hipMemcpyDeviceToHost));
  // printf("d_rows = %d, h_rows = %d, d_cols = %d, h_cols = %d\n", rows, h_rows, cols, h_columns);
  if (rows != this->h_rows || cols != this->h_columns) {
    printf("INCONSISTENT ROWS AND COLS BETWEEN HOST AND DEVICE\n");
  }
  Vector_CPU v_cpu(this->h_rows, this->h_columns, out);
  return v_cpu;
};

double Vector_CUBLAS::Dnrm2() {
  double h_out;
  int *version;
  double *d_out;
  double zero = 0.0;
  gpuErrchk(hipMalloc(&d_out, sizeof(double)));
  int size = (this->h_rows * this->h_columns);

  int incre = 1;
  // stat = hipblasSetVector(size, sizeof(double), (void **)&c_d_mat, incre, d_mat, incre);
  // if (stat != HIPBLAS_STATUS_SUCCESS) {
  //   printf ("CUBLAS Version failed\n");
  //   return EXIT_FAILURE;
  // }

  hipblasStatus_t stat3 = hipblasDnrm2(handle, size, this->d_mat, incre, &h_out);
  gpuErrchk(hipDeviceSynchronize());
  if (stat3 != HIPBLAS_STATUS_SUCCESS) {
    printf("CUBLAS DNRM2 failed\n");
    // return EXIT_FAILURE;
  }

  // stat = hipblasDestroy(handle);
  // if (stat != HIPBLAS_STATUS_SUCCESS) {
  //   printf ("CUBLAS destroy failed\n");
  //   return EXIT_FAILURE;
  // }
  // hipblasDestroy(handle);
  // gpuErrchk(hipMemcpy(&h_out, d_out, sizeof(double), hipMemcpyDeviceToHost));
  assert(!(h_out != h_out));
  return h_out;
};

void Vector_CUBLAS::printmat() {
  unsigned int blocksX = (this->h_rows / 16) + 1;
  unsigned int blocksY = (this->h_columns / 16) + 1;
  dim3 grid(blocksX, blocksY, 1);
  dim3 block(16, 16, 1);

  // print<<<grid, block>>>(this->d_mat, this->d_rows, this->d_columns);
  // gpuErrchk(hipPeekAtLastError());
  hipDeviceSynchronize();
}

Vector_CUBLAS Vector_CUBLAS::transpose() {
  Vector_CUBLAS out(this->h_columns, this->h_rows);
  int m = this->h_rows, n = this->h_columns, k = this->h_columns;
  int lda = m, ldb = k, ldc = m;
  stat = hipblasDgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, m, n, &ZERO, this->d_mat, lda, &ONE, this->d_mat, ldb, out.d_mat, ldc); // ADD-SUB-TRANSPOSE
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    std::cout << cublasGetErrorString(stat) << std::endl;
    printf("CUBLAS matrix transpose failed\n");
  }
  return out;
};