#ifndef CAFFE_COMMON_CUH_
#define CAFFE_COMMON_CUH_
#include <hip/hip_runtime.h>
#include "hipblas.h"

hipError_t cudaStat;
hipblasHandle_t handle;
hipStream_t stream;
hipblasStatus_t stat1 = hipblasCreate(&handle);
hipError_t cudaStat1 = hipStreamCreate(&stream);
hipblasStatus_t stat = hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);
hipblasStatus_t stat2 = hipblasSetStream(handle, stream);
void cublasReset() {
  stat1 = hipblasCreate(&handle);
  cudaStat1 = hipStreamCreate(&stream);
  stat = hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);
  stat2 = hipblasSetStream(handle, stream);
}

// Operator overloads

// Vector_GPU Vector_GPU::operator*(Vector_GPU &v) {
//   hipblasLtMatmul(hipblasLtHandle_t               lightHandle,
//     hipblasLtMatmulDesc_t           computeDesc,
//     const void                    *alpha,
//     const void                    *A,
//     hipblasLtMatrixLayout_t         Adesc,
//     const void                    *B,
//     hipblasLtMatrixLayout_t         Bdesc,
//     const void                    *beta,
//     const void                    *C,
//     hipblasLtMatrixLayout_t         Cdesc,
//     void                          *D,
//     hipblasLtMatrixLayout_t         Ddesc,
//     const hipblasLtMatmulAlgo_t    *algo,
//     void                          *workspace,
//     size_t                         workspaceSizeInBytes,
//     hipStream_t                   stream);
// }

double Vector_GPU::Dnrm2() {
  // int blockX = ((this->h_rows * this->h_columns + TILE_DIM_X - 1) / TILE_DIM_X);
  // dim3 blocks(blockX, 1);
  double h_out;
  int *version;
  double *d_out;
  double zero = 0.0;
  // double c_d_mat[h_rows*h_columns];
  int incre = 1;
  gpuErrchk(hipMalloc(&d_out, sizeof(double)));
  // gpuErrchk(hipMemcpy(d_out, &zero, sizeof(double), hipMemcpyHostToDevice));
  // gpuErrchk(hipMemcpy(&c_d_mat,d_mat, sizeof(double)*h_rows*h_columns,hipMemcpyDeviceToHost));

  // for (int i = 0; i<100;++i){
  //  printf("%f\n",c_d_mat[i]);
  //}
  // gpuErrchk(hipMalloc(&c_d_mat, sizeof(double)*h_rows*h_columns));
  // gpuErrchk(hipMemcpy(&c_d_mat, d_mat, sizeof(double) * h_columns * h_rows, hipMemcpyDeviceToDevice));
  int size = (this->h_rows * this->h_columns);

  // stat = hipblasSetVector(size, sizeof(double), (void **)&c_d_mat, incre, d_d_mat, incre);
  // if (stat != HIPBLAS_STATUS_SUCCESS) {
  //   printf ("CUBLAS Version failed\n");
  //   return EXIT_FAILURE;
  // }

  hipblasStatus_t stat3 = hipblasDnrm2(handle, size, this->d_mat, incre, d_out);
  hipDeviceSynchronize();
  if (stat3 != HIPBLAS_STATUS_SUCCESS) {
    printf("CUBLAS DNRM2 failed\n");
    return EXIT_FAILURE;
  }

  // stat = hipblasDestroy(handle);
  // if (stat != HIPBLAS_STATUS_SUCCESS) {
  //   printf ("CUBLAS destroy failed\n");
  //   return EXIT_FAILURE;
  // }
  // hipblasDestroy(handle);
  gpuErrchk(hipMemcpy(&h_out, d_out, sizeof(double), hipMemcpyDeviceToHost));
  assert(!(h_out != h_out));
  return h_out;
};