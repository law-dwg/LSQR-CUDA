#include "hip/hip_runtime.h"
#include "matVec_gpu.cuh"
//#include "../cpu/matVec_cpu.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <stdio.h>  //NULL, printf
#include <stdlib.h> //srand, rand
#include <string.h>
#include <time.h>

#include <iostream>
#include <sstream>

#include ""
void __global__ print() {}

int main() {
  // Check Cuda Capabale Device
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  int device;

  if (deviceCount > 0) {
    for (device = 0; device < deviceCount; ++device) {
      hipDeviceProp_t deviceProp;
      hipGetDeviceProperties(&deviceProp, device);
      printf("Device %s has compute capability %d.%d.\n", deviceProp.name, deviceProp.major,
             deviceProp.minor);
      printf("Number of multiprocessors: %d\n", deviceProp.multiProcessorCount);
      printf("Clock rate: %d Hz\n", deviceProp.clockRate);
      printf("Total amount of global memory: %d KB\n", deviceProp.totalGlobalMem / 1024);
      printf("Total amount of constant memory: %d KB\n", deviceProp.totalConstMem / 1024);
      printf("Total amount of shared memory per block: %d KB\n",
             deviceProp.sharedMemPerBlock / 1024);
      printf("Total amount of shared memory per SM: %d KB\n", 64);
      printf("Warp size: %d\n", deviceProp.warpSize);
      printf("Maximum number of threads per block: %d\n", deviceProp.maxThreadsPerBlock);
      printf("Maximum number of blocks per multiprocessor: %d\n",
             deviceProp.maxThreadsPerMultiProcessor / deviceProp.maxThreadsPerBlock);
      printf("Maximum number of threads per multiprocessor: %d\n",
             deviceProp.maxThreadsPerMultiProcessor);
      printf("Maximum number of warps per multiprocessor: %d\n",
             deviceProp.maxThreadsPerMultiProcessor / 32);
      printf("Maximum Grid size: (%d,%d,%d)\n", deviceProp.maxGridSize[0],
             deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
      printf("Maximum block dimension: (%d,%d,%d)\n", deviceProp.maxThreadsDim[0],
             deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
    }
    for (int e = 11200; e < (11201); e += 100) {
      unsigned int rows = e;
      unsigned int columns = e;
      int array_size = rows * columns;
      int byte_size = sizeof(double) * array_size;
      double *h_in1 = new double[array_size];
      // double *h_in2 = new double[array_size];
      // double *h_in3 = new double[array_size / 2];
      // double *h_out = new double[array_size * array_size];

      for (int i = 0; i < array_size; i++) {
        h_in1[i] = i;
        // h_in2[i] = 5 * i;
      }
      // for (int i = 0; i < array_size / 2; i++) {
      //   h_in3[i] = 3 * i;
      // }
      Vector_GPU d_i1(rows, columns, h_in1);
      // Vector_GPU d_i2(columns, rows, h_in2);
      hipDeviceSynchronize();
      d_i1 = d_i1.transpose();
      Vector_CPU hd_i1 = d_i1.matDeviceToHost();
      Vector_CPU comparator(rows, columns, h_in1);
      comparator = comparator.transpose();
      double *matGpu = new double[hd_i1.rows * hd_i1.columns];
      double *matCpu = new double[hd_i1.rows * hd_i1.columns];
      matGpu = &hd_i1.mat[0];
      matCpu = &comparator.mat[0];
      bool same = true;
      double epsilon = 0.001;
      do {
        for (int i = 0; i < rows * columns; i++) {
          // printf("matGpu[%d] = %f, matCpu[%d] = %f\n", i, matGpu[i], i, matCpu[i]);
          // printf("DIFF = %f, %f == %f\n", std::abs(matGpu[i] - matCpu[i]), matGpu[i], matCpu[i]);
          if (!(std::abs(matGpu[i] - matCpu[i]) < epsilon)) {
            printf("MATRICIES SIZE (%d x %d) DO NOT MATCH DISCREPANCY AT INDEX %d; DIFF = %f, %f "
                   "== %f\n",
                   rows, columns, i, std::abs(matGpu[i] - matCpu[i]), matGpu[i], matCpu[i]);
            printf("matGpu[%d]=%f, matCpu[%d]=%f\nmatGpu[%d]=%f, matCpu[%d]=%f\nmatGpu[%d]=%f, "
                   "matCpu[%d]=%f\nmatGpu[%d]=%f, matCpu[%d]=%f\n",
                   i - 1, matGpu[i - 1], i - 1, matCpu[i - 1], i, matGpu[i], i, matCpu[i], i + 1,
                   matGpu[i + 1], i + 1, matCpu[i + 1], i + 2, matGpu[i + 2], i + 2, matCpu[i + 2]);
            printf("LAST ELEMENTS matGpu[%d]=%f, matCpu[%d]=%f\n", (rows * columns) - 1,
                   matGpu[(rows * columns) - 1], (rows * columns) - 1,
                   matCpu[(rows * columns) - 1]);
            same = false;
            break;
          }
        }
        if (same) {
          printf("MATRICES MATCH FOR (%d x %d)\n", rows, columns);
          same = false;
        };
      } while (same);
      // comparator.~Vector_CPU();
      // hd_i1.~Vector_CPU();
      delete h_in1, matCpu, matGpu; //, h_in3, h_out;
    }
    hipError_t err = hipGetLastError(); // add
    if (err != hipSuccess) {
      std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl;
    } // add
    hipProfilerStop();
  } else {
    printf("NO CUDA DEVICE AVAILABLE");
  }
  return 0;
}