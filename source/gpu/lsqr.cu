#include "hip/hip_runtime.h"
#include "../cpu/lsqr_cpu.hpp"
#include "../cpu/matVec_cpu.hpp"
#include "../cpu/matrixBuilder.hpp"
// #include "lsqr_gpu.cuh"
#include "matVec_gpu.cuh"
#include <cassert>
#include <chrono>
#include <ctime>
#include <ctype.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <filesystem>
#include <iostream>
#include <math.h>
#include <set>
#include <sstream>
#include <stdio.h>
#include <string>
#include <time.h>
#include <vector>

namespace fs = std::filesystem;

int checkDevice() {
  // Check Cuda Capabale Device
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  int device;

  if (deviceCount > 0) {
    for (device = 0; device < deviceCount; ++device) {
      hipDeviceProp_t deviceProp;
      hipGetDeviceProperties(&deviceProp, device);
      printf("Device %s has compute capability %d.%d.\n", deviceProp.name, deviceProp.major, deviceProp.minor);
      printf("Number of multiprocessors: %d\n", deviceProp.multiProcessorCount);
      printf("Clock rate: %d Hz\n", deviceProp.clockRate);
      printf("Total amount of global memory: %d KB\n", deviceProp.totalGlobalMem / 1024);
      printf("Total amount of constant memory: %d KB\n", deviceProp.totalConstMem / 1024);
      printf("Total amount of shared memory per block: %d KB\n", deviceProp.sharedMemPerBlock / 1024);
      printf("Total amount of shared memory per SM: %d KB\n", 64);
      printf("Warp size: %d\n", deviceProp.warpSize);
      printf("Maximum number of threads per block: %d\n", deviceProp.maxThreadsPerBlock);
      printf("Maximum number of blocks per multiprocessor: %d\n", deviceProp.maxThreadsPerMultiProcessor / deviceProp.maxThreadsPerBlock);
      printf("Maximum number of threads per multiprocessor: %d\n", deviceProp.maxThreadsPerMultiProcessor);
      printf("Maximum number of warps per multiprocessor: %d\n", deviceProp.maxThreadsPerMultiProcessor / 32);
      printf("Maximum Grid size: (%d,%d,%d)\n", deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
      printf("Maximum block dimension: (%d,%d,%d)\n", deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
    }
  } else {
    printf("NO CUDA DEVICE AVAILABLE");
  }
  return deviceCount;
};

template <typename Vec> Vec lsqr(Vec &A, Vec &b) {
  // Iteration
  unsigned int istop = 0;
  unsigned int itn = 0;
  double rho = 0;
  double phi = 0;
  double c = 0;
  double s = 0;
  double theta = 0;
  double tau = 0;
  double res = 0;
  double res1 = 0;
  double ddnorm = 0;
  double Anorm = 0;
  double Acond = 0;
  double damp = 0;
  double dnorm = 0;
  double dknorm = 0;
  double res2 = 0;
  double xnorm = 0;
  double xxnorm = 0;
  double z = 0;
  double sn2 = 0;
  double ctol = 0;
  double rtol = 0;
  double dampsq = 0;
  double cs2 = -1;
  double atol = 1e-8;
  double btol = 1e-8;
  double conlim = 1e8;
  double res_old = 1e10;
  double epsilon = 1e-16;
  double alpha = 0;
  double beta = 0;
  Vec u, v, w, res_v;
  if (conlim > 0) {
    ctol = 1 / conlim;
  };
  dampsq = damp * damp;

  /*1. Initialize*/
  Vec x(A.getColumns(), 1);
  beta = b.Dnrm2();
  // std::cout << beta << std::endl;
  if (beta > 0) {
    u = b * (1 / beta);
    v = A.transpose() * u;
    alpha = v.Dnrm2();
  } else {
    v = x;
    alpha = 0;
  };
  // std::cout << alpha << std::endl;
  // v.print();

  if (alpha > 0) {
    v = v * (1 / alpha);
    w = v;
  };
  double rhobar = alpha;
  double phibar = beta;

  // Norms
  double rnorm = beta;
  double r1norm = rnorm;
  double r2norm = rnorm;
  double Arnorm = alpha * beta;

  // 2. For i=1,2,3....
  printf("2. For i=1,2,3....\n");
  do {
    if (itn == A.getRows() / 2 || itn == A.getRows() || itn == (A.getRows() + A.getRows() / 2))
      printf("itn = %d\n", itn);
    itn++;
    Vec A_T = A.transpose();

    // 3. Continue the bidiagonialization
    /* Important equations for understanding.
    ubar_i+1 = beta_i+1 * u_i+1 = (A*v_i) - (alpha_i*u_i)
    beta_i+1 = ||ubar_i+1||
    u_i+1 =  ubar_i+1 * (1/beta_i+1)
    vbar_i+1 = alpha_i+1 * v_i+1 = (A_t*u_i+1) - (beta_i*v_i)
    alpha_i+1 = ||vbar_i+1||
    v_i+1 =  vbar_i+1 * (1/alpha_i+1)
    */
    // printf("3. Continue the bidiagonialization\n");
    u = A * v - u * alpha; // ubar_i+1

    beta = u.Dnrm2(); // beta_i+1 = ||ubar_i+1||
    if (beta > 0) {
      u = u * (1 / beta);         // u_i+1
      v = (A_T * u) - (v * beta); // vbar_i+1
      alpha = v.Dnrm2();          // alpha_i+1
      if (alpha > 0) {
        v = v * (1 / alpha); // v_i+1
      }
    }

    // 4. Construct and apply next orthogonal transformation
    // printf("4. Construct and apply next orthogonal transformation\n");

    rho = D2Norm(rhobar, beta); // rho_i
    c = rhobar / rho;           // c_i
    s = beta / rho;             // s_i
    theta = s * alpha;          // theta_i+1
    rhobar = -c * alpha;        // rhobar_i+1
    phi = c * phibar;           // phi_i = c_i*phibar_i
    phibar = s * phibar;        // phibar_i+1 = s_i*phibar_i

    // used for stopping critera
    tau = s * phi;
    Arnorm = alpha * std::abs(tau);

    // 5. Update x,w
    // save values for stopping criteria
    Vec dk = w * (1 / rho);
    double dknrm2 = dk.Dnrm2();
    dknorm = dknrm2 * dknrm2 + ddnorm;

    /* Important equations
    x_i = x_i-1 + (phi_i/rho_i) *w_i
    w_i+1 = v_i+1 - (theta_i+1/rho_i)*w_i
    */
    x = x + w * (phi / rho);
    w = v - (w * (theta / rho));
    // residual
    res_v = b - (A * x);
    res = res_v.Dnrm2();

    // 6. Test for convergence
    // printf("6. Test for convergence\n");
    /*Test 1 for convergence
    stop if ||r|| =< btol*||b|| + atol*||A||*||x||
    */
    if (res <= (btol * b.Dnrm2() + atol * A.Dnrm2() * x.Dnrm2())) {
      istop = 1;
    }

    /*Test 2 for convergence
    stop if ||A_T*r||/||A||*||r|| <= atol
    */
    if (Arnorm / (A.Dnrm2() * res) <= atol) {
      istop = 2;
    }

    /*Test 3 for convergence
    stop if cond(A) => conlim

    Acond = A.Dnrm2() * sqrt(ddnorm);
    if (Acond < conlim){
        istop=3;
    }

    res1 = phibar*phibar;
    double test3 = 1/ (Acond + epsilon);
    std::cout.precision(25);
    std::cout<<std::fixed<<res<<std::endl;

    if(res <= btol * b.Dnrm2() + atol * A.Dnrm2() * x.Dnrm2());

    if(test3<=ctol){
        istop=3;
        printf("%i\n",itn);
    };
    double test2 = Arnorm / (A.Dnrm2()*res + epsilon);
    if( 1 + test2 <= 1){
        istop=5;
        printf("%i\n",itn);
    };*/

    // printf("iteration %i\n", itn);
    // printf("istop %i\n", istop);
    // x.print();
    // printf("%f\n", Arnorm);
  } while (istop == 0 && itn < 2 * A.getRows());
  printf("ran through %d iterations \n", itn);
  return x;
}

int main() {
  double sp;
  std::string userName;
  std::cout << "Welcome to law-dwg's lsqr cuda and cpp implementations!\nYou can use ctrl+c to kill this program at any time.\n\nBefore we begin, "
               "please type in your name: ";
  // std::cin >> std::ws; // skip leading whitespace
  // std::getline(std::cin, userName);
  // start
  std::cout << "Hello " << userName << ", Would you like to build the test matrices from scratch? (y/n): ";
  // bool matBuild = yesNo();
  bool matBuild = true;

  if (matBuild) { // build matrices
    std::cout << "\nGreat, lets get started\n\nWhat sparsity should matrix A have? Please enter a number between 0.0-1.0: ";
    // sp = valInput<double>(0.0, 1.0);
    sp = 0;
    std::cout << "Building A Matrices of sparsity " << sp << "\n";
    for (int i = 1000; i < 1100; i += 100) {
      matrixBuilder(i, i, sp, "input/", "A");
      matrixBuilder(i, 1, 0, "input/", "b");
    }
  }

  std::string path_name = "input/";
  std::set<fs::path> sorted_by_name;
  for (auto &entry : fs::directory_iterator(path_name)) // alphabetical listing of files in input
    sorted_by_name.insert(entry.path());

  if (sorted_by_name.size() == 0) { // empty input folder
    std::cout << "Looks like there are no files in the input folder. Please add your own matricies in \"NumOfRows_NumOfCols_A.txt\" and "
                 "\"NumOfRows_1_b.txt\" format, or rerun the "
                 "program to autobuild matrices\n"
              << std::endl;
    return 0;
  };
  checkDevice();
  std::set<fs::path>::iterator it = sorted_by_name.begin();
  while (it != sorted_by_name.end()) { // iterate through sorted files
    std::string file1, file2;
    file1 = *it;
    ++it;
    file2 = *it;
    ++it; // iterate every two files
    std::vector<std::string> files{file1, file2};
    unsigned A_rows, A_cols, b_rows, b_cols;
    std::vector<double> A, b;
    for (auto file : files) {
      fileParserLoader(file, A_rows, A_cols, A, b_rows, b_cols, b);
    }
    bool A_sizecheck, b_sizecheck, Ab_rowscheck, b_colscheck, all_checks;
    A_sizecheck = A.size() == A_rows * A_cols && A_rows != 0 && A_cols != 0;
    b_sizecheck = b.size() == b_rows * b_cols && b_rows != 0 && b_cols == 1;
    Ab_rowscheck = A_rows == b_rows;
    all_checks = A_sizecheck && b_sizecheck && Ab_rowscheck;
    assert(all_checks);
    // if (all_checks) {
    //   continue;
    // } else {
    //   printf("Error, please check the matrix file naming convention (\"NumOfRows_NumOfCols_A.txt\" and "
    //          "\"NumOfRows_1_b.txt\" format) and make sure the naming convention (rows * columns) matches the number of values in each file\n");
    //   return 0;
    // }
    printf("Running lsqr-CPU implementation\nAx=b where A(%d,%d) and b(%d,1)\n", A_rows, A_cols, b_rows);
    Vector_CPU A_c(A_rows, A_cols, A.data());
    Vector_CPU b_c(b_rows, b_cols, b.data());
    // A_c.print();
    // b_c.print();
    std::clock_t c_start = std::clock();
    Vector_CPU x_c = lsqr<Vector_CPU>(A_c, b_c);
    std::clock_t c_end = std::clock();
    long double time_elapsed_ms = 1000.0 * (c_end - c_start) / CLOCKS_PER_SEC;
    std::cout << "CPU time used = " << time_elapsed_ms << " ms for lsqr\n";
    std::string file_out = "output/" + std::to_string(A_cols) + "_1_x_CPU.txt";
    writeArrayToFile(file_out, x_c.getRows(), x_c.getColumns(), x_c.getMat());
    printf("---------------------------------------------\n");
    printf("Running lsqr-GPU implementation\nAx=b where A(%d,%d) and b(%d,1)\n", A_rows, A_cols, b_rows);
    Vector_GPU A_g(A_rows, A_cols, A.data());
    Vector_GPU b_g(b_rows, b_cols, b.data());
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    Vector_GPU x_g = lsqr<Vector_GPU>(A_g, b_g);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("GPU time used = %f ms for lsqr\n", milliseconds);
    file_out = "output/" + std::to_string(A_cols) + "_1_x_GPU.txt";
    Vector_CPU x_g_out = x_g.matDeviceToHost();
    writeArrayToFile(file_out, x_g_out.getRows(), x_g_out.getColumns(), x_g_out.getMat());
    hipblasDestroy(handle);
    hipDeviceReset();
    cublasReset();
  }
  hipblasDestroy(handle);
}