#include "matVec_gpu.cuh"
//#include "../cpu/matVec_cpu.h"
#include ""
#include "lsqr_gpu.cuh"
#include <assert.h>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <limits>
#include <sstream>
#include <stdio.h>  //NULL, printf
#include <stdlib.h> //srand, rand
#include <string.h>
#include <time.h>

int main() {
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  checkDevice();
  unsigned int rows1 = 7; // 167 correct 168 wrong
  unsigned int columns1 = 1;
  int array_size_1 = rows1 * columns1;
  int byte_size_1 = sizeof(double) * array_size_1;
  double *h_in1 = new double[array_size_1];

  for (int i = 0; i < array_size_1; i++) {
    // h_in1[i] = double(rand() % 10 + 1);
    h_in1[i] = double(i);
  }

  Vector_GPU h1(rows1, columns1, h_in1);
  double out = h1.dDnrm2();

  Vector_CPU h2 = h1.matDeviceToHost();
  double out2 = h2.Dnrm2();
  typedef std::numeric_limits<double> dbl;
  printf("GPU: %20f\n", out);
  printf("CPU: %20f\n", out2);
  std::cout.precision(dbl::max_digits10);
  std::cout << out << std::endl;
  std::cout << out2 << std::endl;
  delete h_in1;                         //, matCpu, matGpu;  //, h_in3, h_out;
  hipError_t err = hipGetLastError(); // add
  if (err != hipSuccess) {
    std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl;
  } // add
  hipProfilerStop();

  return 0;
};