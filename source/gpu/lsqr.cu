#include "matVec_gpu.cuh"
//#include "../cpu/matVec_cpu.h"
#include ""
#include "lsqr_gpu.cuh"
#include <assert.h>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <sstream>
#include <stdio.h>  //NULL, printf
#include <stdlib.h> //srand, rand
#include <string.h>
#include <time.h>

int main() {
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  unsigned int rows1 = 267;
  unsigned int columns1 = 2000;
  int array_size_1 = rows1 * columns1;
  int byte_size_1 = sizeof(double) * array_size_1;
  double *h_in1 = new double[array_size_1];

  srand(time(NULL));
  for (int i = 0; i < array_size_1; i++) {
    // h_in1[i] = double(rand() % 10 + 1);
    h_in1[i] = double(i);
  }
  unsigned int rows2 = 2000;
  unsigned int columns2 = 983;
  int array_size_2 = rows2 * columns2;
  int byte_size_2 = sizeof(double) * array_size_2;
  double *h_in2 = new double[array_size_2];
  for (int i = 0; i < array_size_2; i++) {
    // h_in2[i] = double(rand() %10 + 1);
    h_in2[i] = double(i);
  }
  Vector_GPU d1(rows1, columns1, h_in1);
  Vector_GPU d2(rows2, columns2, h_in2);
  Matrix_GPU MD1(10, 10, 0.01);
  Matrix_GPU MD2(10, 10, 0.99);
  Matrix_GPU MD3 = MD1 * MD2;
  Vector_CPU MC1 = MD1.matDeviceToHost();
  Vector_CPU MC2 = MD2.matDeviceToHost();
  Vector_CPU MC3 = MD3.matDeviceToHost();
  //// Vector_CPU MCC3 = MC1 * MC2;
  MC1.print();
  MC2.print();
  MC3.print();
  /// MCC3.print();
  /// compareMat(MCC3.getHMat(), MCC3.getRows(), MCC3.getColumns(), MC3.getHMat(), MC3.getRows(), MC3.getColumns());
  hipDeviceSynchronize();
  // hipEventRecord(start);
  // Vector_GPU d3 = d1 * d2;
  // hipEventRecord(stop);
  // Vector_CPU hd3 = d3.matDeviceToHost();
  // Vector_GPU d4 = d1.multNai(d2);
  // hipEventSynchronize(stop);
  // float milisecs = 0;
  // hipEventElapsedTime(&milisecs, start, stop);
  // printf("GPU %f milliseconds elapsed\n", milisecs);
  // double *matGpu = new double[hd3.rows * hd3.columns];
  // matGpu = &hd3.mat[0];
  // Vector_CPU h1(rows1, columns1, h_in1);
  // Vector_CPU h2(rows2, columns2, h_in2);
  // Vector_CPU h3 = h1 * h2;
  // double *matCpu = new double[h3.rows * h3.columns];
  // matCpu = &h3.mat[0];
  // printf("CHECKING\n");
  // compareMat(matCpu, h3.getRows(), h3.getColumns(), matGpu, d3.getRows(), d3.getColumns());
  delete h_in1, h_in2;                  //, matCpu, matGpu;  //, h_in3, h_out;
  hipError_t err = hipGetLastError(); // add
  if (err != hipSuccess) {
    std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl;
  } // add
  hipProfilerStop();

  return 0;
};