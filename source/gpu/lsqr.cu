#include "hip/hip_runtime.h"
#include "matVec_gpu.cuh"
//#include "../cpu/matVec_cpu.h"
#include "../cpu/lsqr_cpu.h"
#include ""
#include "lsqr_gpu.cuh"
#include <assert.h>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <limits>
#include <sstream>
#include <stdio.h>  //NULL, printf
#include <stdlib.h> //srand, rand
#include <string.h>
#include <time.h>

int main() {
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  checkDevice();

  double *a_heap = new double[3 * 3]{1.0, -2.2, 0, 3.2, 4.0, 0, 5.0, 0, 6.1};
  double *b_heap = new double[3 * 1]{0.01, 0.2, -0.2};
  Vector_CPU A_c(3, 3, a_heap);
  Vector_CPU b_c(3, 1, b_heap);
  A_c.print();
  b_c.print();
  Vector_GPU A = A_c;
  Vector_GPU b = b_c;

  unsigned int istop, itn = 0;

  double ddnorm, Anorm, Acond, damp, dnorm, dknorm, res2, xnorm, xxnorm, z, sn2, rtol = 0;
  double rho, phi, c, s, theta, tau, res, res1;
  double cs2 = -1;
  double atol, btol = 1e-8;

  double conlim = 1e8;
  double ctol = 0;
  if (conlim > 0) {
    ctol = 1 / conlim;
  };

  double dampsq = damp * damp;

  double res_old = 1e10;
  double epsilon = 1e-16;

  /*1. Initialize*/
  Vector_GPU u, v, w, res_v;
  double alpha;
  Vector_GPU x(A.getColumns(), 1);
  double beta = b.Dnrm2();
  std::cout << beta << std::endl;
  if (beta > 0) {
    u = b * (1 / beta);
    v = A.transpose() * u;
    alpha = v.Dnrm2();
  } else {
    v = x;
    alpha = 0;
  };
  std::cout << alpha << std::endl;
  v.printmat();

  if (alpha > 0) {
    v = v * (1 / alpha);
    w = v;
  };
  double rhobar = alpha;
  double phibar = beta;

  // Norms
  double rnorm = beta;
  double r1norm = rnorm;
  double r2norm = rnorm;
  double Arnorm = alpha * beta;

  // 2. For i=1,2,3....
  printf("2. For i=1,2,3....\n");
  do {
    itn++;
    Vector_GPU A_T = A.transpose();

    // 3. Continue the bidiagonialization
    /* Important equations for understanding.
    ubar_i+1 = beta_i+1 * u_i+1 = (A*v_i) - (alpha_i*u_i)
    beta_i+1 = ||ubar_i+1||
    u_i+1 =  ubar_i+1 * (1/beta_i+1)
    vbar_i+1 = alpha_i+1 * v_i+1 = (A_t*u_i+1) - (beta_i*v_i)
    alpha_i+1 = ||vbar_i+1||
    v_i+1 =  vbar_i+1 * (1/alpha_i+1)
    */
    printf("3. Continue the bidiagonialization\n");
    u = A * v - u * alpha; // ubar_i+1
    beta = u.Dnrm2();      // beta_i+1 = ||ubar_i+1||
    if (beta > 0) {
      u = u * (1 / beta);         // u_i+1
      v = (A_T * u) - (v * beta); // vbar_i+1
      alpha = v.Dnrm2();          // alpha_i+1
      if (alpha > 0) {
        v = v * (1 / alpha); // v_i+1
      }
    }

    // 4. Construct and apply next orthogonal transformation
    printf("4. Construct and apply next orthogonal transformation\n");

    rho = D2Norm(rhobar, beta); // rho_i
    c = rhobar / rho;           // c_i
    s = beta / rho;             // s_i
    theta = s * alpha;          // theta_i+1
    rhobar = -c * alpha;        // rhobar_i+1
    phi = c * phibar;           // phi_i = c_i*phibar_i
    phibar = s * phibar;        // phibar_i+1 = s_i*phibar_i

    // used for stopping critera
    tau = s * phi;
    Arnorm = alpha * std::abs(tau);

    // 5. Update x,w
    printf("5. Update x,w\n");
    // save values for stopping criteria
    Vector_GPU dk = w * (1 / rho);
    dknorm = dk.Dnrm2() * dk.Dnrm2() + ddnorm;
    /* Important equations
    x_i = x_i-1 + (phi_i/rho_i) *w_i
    w_i+1 = v_i+1 - (theta_i+1/rho_i)*w_i
    */
    x = x + w * (phi / rho);
    w = v - (w * (theta / rho));
    // residual
    res_v = b - A * x;
    res = res_v.Dnrm2();

    // 6. Test for convergence
    printf("6. Test for convergence\n");
    /*Test 1 for convergence
    stop if ||r|| =< btol*||b|| + atol*||A||*||x||
    */
    if (res <= (btol * b.Dnrm2() + atol * A.Dnrm2() * x.Dnrm2())) {
      istop = 1;
    }

    /*Test 2 for convergence
    stop if ||A_T*r||/||A||*||r|| <= atol
    */
    if (Arnorm / (A.Dnrm2() * res) <= atol) {
      istop = 2;
    }

    /*Test 3 for convergence
    stop if cond(A) => conlim

    Acond = A.Dnrm2() * sqrt(ddnorm);
    if (Acond < conlim){
        istop=3;
    }

    res1 = phibar*phibar;
    double test3 = 1/ (Acond + epsilon);
    std::cout.precision(25);
    std::cout<<std::fixed<<res<<std::endl;

    if(res <= btol * b.Dnrm2() + atol * A.Dnrm2() * x.Dnrm2());

    if(test3<=ctol){
        istop=3;
        printf("%i\n",itn);
    };
    double test2 = Arnorm / (A.Dnrm2()*res + epsilon);
    if( 1 + test2 <= 1){
        istop=5;
        printf("%i\n",itn);
    };*/
    printf("iteration %i\n", itn);
    printf("istop %i\n", istop);
    x.printmat();
    printf("%f\n", Arnorm);
  } while (istop == 0);
  delete a_heap, b_heap;
  hipError_t err = hipGetLastError(); // add
  if (err != hipSuccess) {
    std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl;
  } // add
  hipProfilerStop();

  return 0;
};