#include "hip/hip_runtime.h"
#include "matVec_gpu.cuh"
#include <stdio.h> //NULL, printf
#include <stdlib.h> //srand, rand
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include ""
#include <sstream>
#include <iostream>
#include <string.h>
#include <time.h>

void __global__ print(){
    
}

int main(){
    //Check Cuda Capabale Device
    int deviceCount;
    hipGetDeviceCount (&deviceCount);
    int device;

    if(deviceCount>0){
        for (device = 0; device < deviceCount; ++device) {
            hipDeviceProp_t deviceProp;
            hipGetDeviceProperties (&deviceProp, device);
            printf ("Device %s has compute capability %d.%d.\n", deviceProp.name, deviceProp.major, deviceProp.minor);
        }
        int array_size = 6;
        int byte_size = sizeof(double) * array_size;
        double *h_in1 = new double [array_size];
        double *h_in2 = new double [array_size];
        double * h_out = new double [array_size*array_size];
        for (int i = 0; i < array_size; i++){
            h_in1[i]=i;
            h_in2[i]=2*i;
        }
        for (int i = 0; i<array_size; i++){
            std::cout<<h_in2[i]<<std::endl;
        }
        Vector_GPU d_i1(array_size,1,h_in1);
        Vector_GPU d_i2(array_size,1,h_in2);
        unsigned int *rows = new unsigned int;
        unsigned int *columns = new unsigned int;
        hipMemcpy(h_in1,d_i2.d_mat,sizeof(double)*array_size*1,hipMemcpyDeviceToHost);
        hipMemcpy(rows,d_i2.d_rows,sizeof(unsigned int),hipMemcpyDeviceToHost);
        hipMemcpy(columns,d_i2.d_columns,sizeof(unsigned int),hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        /*Vector_GPU d_out = d_i1 * d_i2;
        hipMemcpy(h_out,d_out.d_mat,byte_size,hipMemcpyDeviceToHost);*/
        for (int i = 0; i<array_size; i++){
            std::cout<<h_in1[i]<<std::endl;
        }
        printf("%d x %d\n",*rows,*columns);
        delete h_in1, h_in2, h_out, rows, columns;

    }
    else{
        printf("NO CUDA DEVICE AVAILABLE");
    }
    return 0;
}